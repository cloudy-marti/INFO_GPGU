#include "hip/hip_runtime.h"
#include "SDL_image.h"
#include <stdio.h>
#include <time.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <>
#include "cuStopwatch.cu"

enum conv_t{
    CONV_IDENTITY,
    CONV_EDGE,
    CONV_SHARP,
    CONV_GAUSS
};

SDL_Window *screen;
SDL_Window *screen_res;
SDL_Renderer *ren;
SDL_Renderer *ren_res;
SDL_Texture *tex;
SDL_Texture *tex_res;
SDL_Surface *surf;

int32_t width, height;
float filter[9];

float __constant__ filter_device[9];

__global__ void conv_global(const unsigned char* src, unsigned char* dest, int32_t w, int32_t h){
    // todo: write a kernel to apply the given filter on the given image stored in the global memory
}

float conv_global_gpu(unsigned char* pixels, int32_t w, int32_t h){
    // todo: write the code that manages memory (global memory) and invokes the kernel conv_global, it should return the running time
}

__global__ void conv_texture(hipTextureObject_t src, unsigned char* dest, int32_t w, int32_t h){
    // todo: write a kernel to apply the given filter on the given image stored as a texture
}

float conv_texture_gpu(unsigned char* pixels, int32_t w, int32_t h){
    // todo: write the code that manages memory (texture memory) and invokes the kernel conv_global, it should return the running time
}

int main(int argc, char** argv){
    SDL_Event event;
    bool withtex = false;
    
    // Initialize SDL
    if( SDL_Init(SDL_INIT_VIDEO) < 0 ) {
        fprintf(stderr, "Couldn't initialize SDL: %s\n", SDL_GetError());
        exit(1);
    }
	atexit(SDL_Quit);
    
    if(argc == 1){
        exit(1);
    }
    
    // Read image and option
    IMG_Init(IMG_INIT_PNG);
    surf = IMG_Load(argv[1]);
    if(surf == NULL){
        fprintf(stderr, "Error loading image.\n");
        exit(1);
    }
    width = surf->w;
    height = surf->h;
    SDL_SetSurfaceRLE(surf, 1);
    
    // Initialize involution kernel
    conv_t conv_type;
    if(argc >= 3){
        if (strcmp(argv[2], "identity") == 0) conv_type = CONV_IDENTITY;
        else if (strcmp(argv[2], "edge") == 0) conv_type= CONV_EDGE;
        else if (strcmp(argv[2], "sharp") == 0) conv_type= CONV_SHARP;
        else if (strcmp(argv[2], "gauss") == 0) conv_type = CONV_GAUSS;
        else conv_type = CONV_IDENTITY;
    }
    switch(conv_type){
        case CONV_EDGE:
            filter[0] = -1; filter[1] = -1; filter[2] = -1; 
            filter[3] = -1; filter[4] = 8; filter[5] = -1; 
            filter[6] = -1; filter[7] = -1; filter[8] = -1; 
            break;
        case CONV_SHARP:
            filter[0] = 0; filter[1] = -1; filter[2] = 0; 
            filter[3] = -1; filter[4] = 5; filter[5] = -1; 
            filter[6] = 0; filter[7] = -1; filter[8] = 0; 
            break;
        case CONV_GAUSS:
            filter[0] = 1.0f/16; filter[1] = 1.0f/8; filter[2] = 1.0f/16; 
            filter[3] = 1.0f/8; filter[4] = 1.0f/4; filter[5] = 1.0f/8; 
            filter[6] = 1.0f/16; filter[7] = 1.0f/8; filter[8] = 1.0f/8; 
            break;
        default:
            filter[0] = 0; filter[1] = 0; filter[2] = 0; 
            filter[3] = 0; filter[4] = 1; filter[5] = 0; 
            filter[6] = 0; filter[7] = 0; filter[8] = 0; 
            break;
    }
    hipMemcpyToSymbolAsync(HIP_SYMBOL(filter_device), filter, sizeof(float)*9, 0, hipMemcpyHostToDevice);
    
    if(argc >= 4){
        if(strcmp(argv[3], "texture") == 0) withtex = true;
    }
    
    // Create window
	screen = SDL_CreateWindow("Original", 
                        100,
                        100,
                        width, height, SDL_WINDOW_SHOWN);
    if ( screen == NULL ) {
        fprintf(stderr, "Couldn't set up window: %s\n", SDL_GetError());
        exit(1);
    }
    screen_res = SDL_CreateWindow("Filtered", 
                        300,
                        300,
                        width, height, SDL_WINDOW_SHOWN);
    if ( screen_res == NULL ) {
        fprintf(stderr, "Couldn't set up window: %s\n", SDL_GetError());
        exit(1);
    }
    
    // Initialize CUDA
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
    
    // Create renderer and texture
    ren = SDL_CreateRenderer(screen, -1, SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);
    tex = SDL_CreateTextureFromSurface(ren, surf);
    
    // Show image
    SDL_RenderCopy(ren, tex, NULL, NULL);
    SDL_RenderPresent(ren);
    
    // Compute
    SDL_LockSurface(surf);
    float elapsed;
    if(withtex){
        elapsed = conv_texture_gpu((unsigned char*)surf->pixels, width, height);
    }else{
        elapsed = conv_global_gpu((unsigned char*)surf->pixels, width, height);
    }
    SDL_UnlockSurface(surf);
    
    // Show computed image
    ren_res = SDL_CreateRenderer(screen_res, -1, SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);
    tex_res = SDL_CreateTextureFromSurface(ren_res, surf);
    SDL_RenderCopy(ren_res, tex_res, NULL, NULL);
    SDL_RenderPresent(ren_res);
    SDL_FreeSurface(surf);
    
    while (1) {
        SDL_WaitEvent(&event);
        if ((event.type == SDL_QUIT) || ((event.type == SDL_WINDOWEVENT) && (event.window.event == SDL_WINDOWEVENT_CLOSE))) break;
    }
    
    char s[100];
    sprintf(s, "Kernel execution time: %.4fms", elapsed);
    SDL_ShowSimpleMessageBox(SDL_MESSAGEBOX_INFORMATION, "Timing", s, screen);
    SDL_DestroyTexture(tex);
    SDL_DestroyRenderer(ren);
    SDL_DestroyWindow(screen);
    SDL_DestroyWindow(screen_res);
    exit(0);
}