#include <hip/hip_runtime.h>


class cuStopwatch{
    // todo: add your internal data structure, all in private
	private:
		hipEvent_t startEvent;
		hipEvent_t endEvent;
		bool started;

    public:
        cuStopwatch();
        ~cuStopwatch();
        void start();
        float stop();
};

cuStopwatch::cuStopwatch(){
    // todo: constructor
    hipEventCreate(&startEvent);
    hipEventCreate(&endEvent);
    started = false;
}

cuStopwatch::~cuStopwatch(){
    // todo: destructor
    hipEventDestroy(startEvent);
    hipEventDestroy(endEvent);
}

void cuStopwatch::start(){
    // todo: start the stopwatch, and ignore double start
    if(!started) {
    	hipEventRecord(startEvent);
    	started = true;
    }
}

float cuStopwatch::stop(){
    // todo: stop the stopwatch and return elapsed time, ignore invalid stops (e.g. stop when not yet started or double stop)
	if(!started) {
		return -1;
	}
	hipEventSynchronize(startEvent);
	
	hipEventRecord(endEvent);
	hipEventSynchronize(endEvent);
	
	float ms;
	hipEventElapsedTime(&ms, startEvent, endEvent);

	started = false;
	
	return ms;
}